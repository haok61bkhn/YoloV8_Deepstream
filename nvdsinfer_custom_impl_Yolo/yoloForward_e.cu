/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

__global__ void gpuYoloLayer_e(
    const float* cls, const float* reg, int* num_detections, float* detection_boxes, float* detection_scores,
    int* detection_classes, const float scoreThreshold, const uint netWidth, const uint netHeight,
    const uint numOutputClasses, const uint64_t outputSize)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_id >= outputSize)
        return;

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = cls[x_id * numOutputClasses + i];

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    if (maxProb < scoreThreshold)
        return;

    int count = (int)atomicAdd(num_detections, 1);

    detection_boxes[count * 4 + 0] = reg[x_id * 4 + 0];
    detection_boxes[count * 4 + 1] = reg[x_id * 4 + 1];
    detection_boxes[count * 4 + 2] = reg[x_id * 4 + 2];
    detection_boxes[count * 4 + 3] = reg[x_id * 4 + 3];
    detection_scores[count] = maxProb;
    detection_classes[count] = maxIndex;
}

hipError_t cudaYoloLayer_e(
    const void* cls, const void* reg, void* num_detections, void* detection_boxes, void* detection_scores,
    void* detection_classes, const uint& batchSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& numOutputClasses, hipStream_t stream);

hipError_t cudaYoloLayer_e(
    const void* cls, const void* reg, void* num_detections, void* detection_boxes, void* detection_scores,
    void* detection_classes, const uint& batchSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& numOutputClasses, hipStream_t stream)
{
    int threads_per_block = 16;
    int number_of_blocks = (outputSize / threads_per_block) + 1;

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayer_e<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(cls) + (batch * numOutputClasses * outputSize),
            reinterpret_cast<const float*>(reg) + (batch * 4 * outputSize),
            reinterpret_cast<int*>(num_detections) + (batch),
            reinterpret_cast<float*>(detection_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<float*>(detection_scores) + (batch * outputSize),
            reinterpret_cast<int*>(detection_classes) + (batch * outputSize),
            scoreThreshold, netWidth, netHeight, numOutputClasses, outputSize);
    }
    return hipGetLastError();
}
