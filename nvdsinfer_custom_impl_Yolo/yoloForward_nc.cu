/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void gpuYoloLayer_nc(
    const float* input, int* num_detections, float* detection_boxes, float* detection_scores, int* detection_classes,
    const float scoreThreshold, const uint netWidth, const uint netHeight, const uint gridSizeX, const uint gridSizeY,
    const uint numOutputClasses, const uint numBBoxes, const float scaleXY, const float* anchors, const int* mask)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
        return;

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    const float objectness
        = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)];

    if (objectness < scoreThreshold)
        return;

    int count = (int)atomicAdd(num_detections, 1);

    const float alpha = scaleXY;
    const float beta = -0.5 * (scaleXY - 1);

    float x
        = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
          * alpha + beta + x_id) * netWidth / gridSizeX;

    float y
        = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
          * alpha + beta + y_id) * netHeight / gridSizeY;

    float w
        = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)] * 2, 2)
          * anchors[mask[z_id] * 2];

    float h
        = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)] * 2, 2)
          * anchors[mask[z_id] * 2 + 1];

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    detection_boxes[count * 4 + 0] = x - 0.5 * w;
    detection_boxes[count * 4 + 1] = y - 0.5 * h;
    detection_boxes[count * 4 + 2] = x + 0.5 * w;
    detection_boxes[count * 4 + 3] = y + 0.5 * h;
    detection_scores[count] = objectness * maxProb;
    detection_classes[count] = maxIndex;
}

hipError_t cudaYoloLayer_nc(
    const void* input, void* num_detections, void* detection_boxes, void* detection_scores, void* detection_classes,
    const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream);

hipError_t cudaYoloLayer_nc(
    const void* input, void* num_detections, void* detection_boxes, void* detection_scores, void* detection_classes,
    const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayer_nc<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * inputSize),
            reinterpret_cast<int*>(num_detections) + (batch),
            reinterpret_cast<float*>(detection_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<float*>(detection_scores) + (batch * outputSize),
            reinterpret_cast<int*>(detection_classes) + (batch * outputSize),
            scoreThreshold, netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes, scaleXY,
            reinterpret_cast<const float*>(anchors), reinterpret_cast<const int*>(mask));
    }
    return hipGetLastError();
}
