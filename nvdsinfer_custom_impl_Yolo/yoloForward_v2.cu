/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__device__ void softmaxGPU(
    const float* input, const int bbindex, const int numGridCells, uint z_id, const uint numOutputClasses, float temp,
    float* output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for (i = 0; i < numOutputClasses; ++i) {
        int val = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
        largest = (val>largest) ? val : largest;
    }
    for (i = 0; i < numOutputClasses; ++i) {
        float e = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] / temp - largest / temp);
        sum += e;
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] = e;
    }
    for (i = 0; i < numOutputClasses; ++i) {
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] /= sum;
    }
}

__global__ void gpuRegionLayer(
    const float* input, float* softmax, int* num_detections, float* detection_boxes, float* detection_scores,
    int* detection_classes, const float scoreThreshold, const uint netWidth, const uint netHeight, const uint gridSizeX,
    const uint gridSizeY, const uint numOutputClasses, const uint numBBoxes, const float* anchors)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
        return;

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    const float objectness
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    if (objectness < scoreThreshold)
        return;

    int count = (int)atomicAdd(num_detections, 1);

    float x
        = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)])
          + x_id) * netWidth / gridSizeX;

    float y
        = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)])
          + y_id) * netHeight / gridSizeY;

    float w
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)])
          * anchors[z_id * 2] * netWidth / gridSizeX;

    float h
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)])
          * anchors[z_id * 2 + 1] * netHeight / gridSizeY;

    softmaxGPU(input, bbindex, numGridCells, z_id, numOutputClasses, 1.0, softmax);

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = softmax[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    detection_boxes[count * 4 + 0] = x - 0.5 * w;
    detection_boxes[count * 4 + 1] = y - 0.5 * h;
    detection_boxes[count * 4 + 2] = x + 0.5 * w;
    detection_boxes[count * 4 + 3] = y + 0.5 * h;
    detection_scores[count] = objectness * maxProb;
    detection_classes[count] = maxIndex;
}

hipError_t cudaRegionLayer(
    const void* input, void* softmax, void* num_detections, void* detection_boxes, void* detection_scores,
    void* detection_classes, const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold,
    const uint& netWidth, const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses,
    const uint& numBBoxes, const void* anchors, hipStream_t stream);

hipError_t cudaRegionLayer(
    const void* input, void* softmax, void* num_detections, void* detection_boxes, void* detection_scores,
    void* detection_classes, const uint& batchSize, uint64_t& inputSize, uint64_t& outputSize, const float& scoreThreshold,
    const uint& netWidth, const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses,
    const uint& numBBoxes, const void* anchors, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuRegionLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * inputSize),
            reinterpret_cast<float*>(softmax) + (batch * inputSize),
            reinterpret_cast<int*>(num_detections) + (batch),
            reinterpret_cast<float*>(detection_boxes) + (batch * 4 * outputSize),
            reinterpret_cast<float*>(detection_scores) + (batch * outputSize),
            reinterpret_cast<int*>(detection_classes) + (batch * outputSize),
            scoreThreshold, netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes,
            reinterpret_cast<const float*>(anchors));
    }
    return hipGetLastError();
}
